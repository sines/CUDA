#include"hip/hip_runtime.h"


#define imin(a,b) (a<b?a:b)

const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock -1) / threadsPerBlock);
__global__ void dot(float *a, float *b, float *c)
{
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;
	
	float temp = 0;
	
	while(tid < N)
	{
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}

	cache[cacheIndex] = temp;
	__syncthreads();

	int i = blockDim.x / 2;
	while(i != 0)
	{
		if(cacheIndex < i)
		{
			cache[cacheIndex] +=  cache[cacheIndex + i];
			__syncthreads();
			i /= 2;
		}
		if(cacheIndex == 0)
		{
			c[blockIdx.x] = cache[0];
		}
	}
}

int main(void)
{
	float *a, *b, c, *partial_c;
	float *dev_a, *dev_b, *dev_partial_c;

	a = (float*)malloc(N*sizeof(float));
	b = (float*)malloc(N*sizeof(float));
	partial_c = (float*)malloc(blocksPerGrid * sizeof(float));
	
}
